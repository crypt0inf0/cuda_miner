#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#include <string.h>
#include "sha256.cuh"

#define SHA256_BLOCK_SIZE 32

/*********************** FUNCTION DEFINITIONS ***********************/
__device__ void uint32_to_little_endian(uint32_t value, unsigned char *buffer) {
    buffer[0] = value & 0xFF;
    buffer[1] = (value >> 8) & 0xFF;
    buffer[2] = (value >> 16) & 0xFF;
    buffer[3] = (value >> 24) & 0xFF;
}

void hexStringToByteArray(const char *hexstr, unsigned char *output) {
    while (*hexstr && hexstr[1]) {
        sscanf(hexstr, "%2hhx", output++);
        hexstr += 2;
    }
}

unsigned char* hexstr_to_char(const char* hexstr) {
    size_t len = strlen(hexstr);
    size_t final_len = len / 2;
    unsigned char* chars = (unsigned char*)malloc((final_len + 1));
    for(size_t i = 0, j = 0; j < final_len; i += 2, j++)
        chars[j] = (hexstr[i] % 32 + 9) % 25 * 16 + (hexstr[i+1] % 32 + 9) % 25;
    chars[final_len] = '\0';
    return chars;
}

void hexstr_to_intarray(const char* hexstr, uint32_t* outputloc) {
    size_t len = strlen(hexstr);
    size_t intlen = (len + 7) / 8;
    unsigned char* bytes = hexstr_to_char(hexstr);

    for(size_t i = 0; i < intlen; i++) {
        *(outputloc + i) = ((uint32_t)bytes[i * 4])
            + ((uint32_t)bytes[i * 4 + 1] << 8)
            + ((uint32_t)bytes[i * 4 + 2] << 16)
            + ((uint32_t)bytes[i * 4 + 3] << 24);
    }
    free(bytes);
}

uint32_t reverse32(uint32_t value) {
    return (((value & 0x000000FF) << 24) |
            ((value & 0x0000FF00) << 8) |
            ((value & 0x00FF0000) >> 8) |
            ((value & 0xFF000000) >> 24));
}

void print_bytes(const unsigned char *data, size_t dataLen, int format) {
    for(size_t i = 0; i < dataLen; ++i) {
        printf("%02x", data[i]);
        if (format) {
            printf(((i + 1) % 16 == 0) ? "\n" : " ");
        }
    }
    printf("\n");
}

void print_bytes_reversed(const unsigned char *data, size_t dataLen, int format) {
    for(size_t i = dataLen; i > 0; --i) {
        printf("%02x", data[i - 1]);
        if (format) {
            printf(((i - 1) % 16 == 0) ? "\n" : " ");
        }
    }
    printf("\n");
}

void setDifficulty(uint32_t bits, uint32_t *difficulty) {
    for(int i = 0; i < 8; i++)
        difficulty[i] = 0;

    bits = reverse32(bits);

    char exponent = bits & 0xff;
    uint32_t significand = bits >> 8;

    for(int i = 0; i < 3; i++) {
        unsigned char thisvalue = (unsigned char)(significand >> (8 * i));
        int index = 32 - exponent + i;
        difficulty[index / 4] = difficulty[index / 4] |
            ((unsigned int)thisvalue << (8 * (3 - (index % 4))));
    }
}

__device__ void hashBlock(uint32_t nonce, BYTE* blockHeader, uint32_t *result) {
    uint32_to_little_endian(nonce, blockHeader + 76);

    BYTE buf[SHA256_BLOCK_SIZE];
    SHA256_CTX ctx;

    sha256_init(&ctx);
    sha256_update(&ctx, blockHeader, 80);
    sha256_final(&ctx, buf);

    sha256_init(&ctx);
    sha256_update(&ctx, buf, SHA256_BLOCK_SIZE);
    sha256_final(&ctx, buf);

    memcpy(result, buf, SHA256_BLOCK_SIZE);
}

__global__ void mine_kernel(uint32_t nonce_start, BYTE* blockHeader, uint32_t *hash) {
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t nonce = nonce_start + tid;

    hashBlock(nonce, blockHeader, hash);
}

void prepare_blockHeader(BYTE* blockHeader, const char* version, const char* prev_block_hash, const char* merkle_root, const char* time, const char* nbits) {
    hexStringToByteArray(version, blockHeader);
    hexStringToByteArray(prev_block_hash, blockHeader + 4);
    hexStringToByteArray(merkle_root, blockHeader + 36);
    hexStringToByteArray(time, blockHeader + 68);
    hexStringToByteArray(nbits, blockHeader + 72);
}

uint32_t mineBlock(uint32_t noncestart, const char *version, const char *prev_block_hash, const char *merkle_root, const char *time, const char *nbits) {
    BYTE *blockHeader;
    uint32_t *hash;
    hipMallocManaged(&blockHeader, 80 * sizeof(BYTE));
    hipMallocManaged(&hash, SHA256_BLOCK_SIZE * sizeof(uint32_t));

    prepare_blockHeader(blockHeader, version, prev_block_hash, merkle_root, time, nbits);

    uint32_t difficulty[8];
    uint32_t bits[1];
    hexstr_to_intarray(nbits, bits);
    setDifficulty(*bits, difficulty);

    clock_t start = clock();

    while (1) {
        int numBlocks = 1;
        int threads_per_block = 1;
        mine_kernel<<<numBlocks, threads_per_block>>>(noncestart, blockHeader, hash);
        hipDeviceSynchronize();

        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
            break;
        }

        for (int i = 0; i < 8; i++) {
            if (hash[7-i] < difficulty[i]) {
                print_bytes_reversed((unsigned char *)hash, 32, 1);
                hipFree(blockHeader);
                hipFree(hash);
                return noncestart;
            } else if (hash[7-i] > difficulty[i]) {
                break;
            }
        }

        noncestart += numBlocks * threads_per_block;

        if (((noncestart - numBlocks * threads_per_block) % 500000) == 0) {
            clock_t end = clock();
            double time_spent = (double)(end - start) / CLOCKS_PER_SEC;
            double hashrate = 500000 / time_spent;

            printf("%f hashes per second\n", hashrate);

            start = clock();
        }
    }

    hipFree(blockHeader);
    hipFree(hash);

    return 0;
}

int main() {
    const char *version = "01000000";
    const char *prev_block_hash = "0000000000000000000000000000000000000000000000000000000000000000";
    const char *merkle_root = "3BA3EDFD7A7B12B27AC72C3E67768F617FC81BC3888A51323A9FB8AA4B1E5E4A";
    const char *time = "29AB5F49";
    const char *nbits = "FFFF001D";

    uint32_t nonce = mineBlock(2083236893, version, prev_block_hash, merkle_root, time, nbits);
    printf("Nonce found: %u\n", nonce);

    return 0;
}
  